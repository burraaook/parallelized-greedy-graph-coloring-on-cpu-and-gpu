#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <fstream>
#include <sstream>
#include <iostream>
#include <unordered_set>
#include <unordered_map>
#include <algorithm>

// atomicCAS
#include <device_atomic_functions.h>

#define TEST_NUM 20

// Define the maximum number of bits for the BitSet
#define BITS_PER_WORD 32
#define MAX_BITS 512
#define INVALID_MAX_SIZE 1000

#define IDLE 0
#define RUNNING 1

// cuda error checking
#define cudaCheckError() {                                          \
    hipError_t e=hipGetLastError();                                 \
    if(e!=hipSuccess) {                                              \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
        exit(0); \
    }                                                                 \
}

// BitSet structure definition
typedef struct {
    uint32_t bits[MAX_BITS / BITS_PER_WORD];
} BitSet;

// adj list graph structure, csr format
struct Graph
{
    int num_nodes;
    int num_edges;
    int* offsets;
    int* adj_list;
};


// Initialize the BitSet to zero
__device__ void bitset_init(BitSet* bs) {
    for (size_t i = 0; i < MAX_BITS / BITS_PER_WORD; ++i) {
        bs->bits[i] = 0;
    }
}

// Set a bit
__device__ void bitset_set(BitSet* bs, size_t index) {
    if (index >= MAX_BITS) return;
    bs->bits[index / BITS_PER_WORD] |= (1U << (index % BITS_PER_WORD));
}

// Clear a bit
__device__ void bitset_clear(BitSet* bs, size_t index) {
    if (index >= MAX_BITS) return;
    bs->bits[index / BITS_PER_WORD] &= ~(1U << (index % BITS_PER_WORD));
}

// Get a bit
__device__ int bitset_get(const BitSet* bs, size_t index) {
    if (index >= MAX_BITS) return 0;
    return (bs->bits[index / BITS_PER_WORD] & (1U << (index % BITS_PER_WORD))) != 0;
}

// Reset all bits
__device__ void bitset_reset(BitSet* bs) {
    for (size_t i = 0; i < MAX_BITS / BITS_PER_WORD; ++i) {
        bs->bits[i] = 0;
    }
}

// Bitwise OR
__device__ void bitset_or(BitSet* dest, const BitSet* src) {
    for (size_t i = 0; i < MAX_BITS / BITS_PER_WORD; ++i) {
        dest->bits[i] |= src->bits[i];
    }
}

// Bitwise NOT
__device__ void bitset_not(BitSet* dest, const BitSet* src) {
    for (size_t i = 0; i < MAX_BITS / BITS_PER_WORD; ++i) {
        dest->bits[i] = ~src->bits[i];
    }
}

// Increment BitSet
__device__ void bitset_increment(BitSet* bs) {
    int carry = 1;
    for (size_t i = 0; i < MAX_BITS / BITS_PER_WORD; ++i) {
        uint64_t temp = (uint64_t)bs->bits[i] + carry;
        bs->bits[i] = temp & 0xFFFFFFFF;
        carry = temp >> 32;
        if (carry == 0) break;
    }
}

__device__ void bitset_assign_one(BitSet* bs) {
    // Reset all bits to 0
    bitset_reset(bs);
    // Set the rightmost bit to 1
    bitset_set(bs, 0);
}

// Helper function to print the BitSet (for debugging)
__device__ void bitset_print(const BitSet* bs) {
    for (int i = MAX_BITS - 1; i >= 0; --i) {
        printf("%d", bitset_get(bs, i));
    }
    printf("\n");
}

// is 0
__device__ bool bitset_is_zero(const BitSet* bs) {
    for (int i = 0; i < MAX_BITS / BITS_PER_WORD; ++i) {
        if (bs->bits[i] != 0) {
            return false;
        }
    }
    return true;
}

// Bitwise AND
__device__ void bitset_and(BitSet* dest, const BitSet* src1, const BitSet* src2) {
    for (size_t i = 0; i < MAX_BITS / BITS_PER_WORD; ++i) {
        dest->bits[i] = src1->bits[i] & src2->bits[i];
    }
}


// kernel function which operates depending on warp
__global__ void kernel(const int* offsets, const int* adj_list, BitSet* result, int* queue, 
    bool* conflict_table, int* warp_state, int* num_vertices, int* num_warps, bool* terminate,
    int* warp_vertex_table)
{
    // get the thread id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // get the warp id
    int wid = tid / 32;
    // get the lane id
    int lane = tid % 32;
    // first warp is the master warp
    if (wid == 0 && lane == 0) {
        
        // cursor for the queue
        int cursor = 0;

        while (true) {
            // if queue is empty, wait for all warps to finish, and then terminate
            if (cursor >= *num_vertices)
            {

                for (int i = 1; i < *num_warps; ++i) {
                    while (warp_state[i] != IDLE)
                    {
                        __threadfence();
                    }
                }

                // set terminate flag to true
                *terminate = true;
                __threadfence();

                break;
            }

            // iterate through the warp table
            for (int i = 1; i < *num_warps; ++i) {
                // get warp id and status
                int state = warp_state[i];
                // if warp is IDLE, assign tasks to the warp
                if (state == IDLE) {

                    // assign vertex
                    warp_vertex_table[i] = queue[cursor];

                    // set dct false
                    conflict_table[queue[cursor]] = false;

                    // increment cursor
                    cursor++;

                    atomicExch(&warp_state[i], RUNNING);
                    __threadfence();
                }

                if (cursor >= *num_vertices) {
                    break;
                }
            }
        }
    }

    // first thread of other warps
    else if (lane == 0) {
        while(true) {
            // wait for the master warp to assign tasks
            while (warp_state[wid] != RUNNING && !(*terminate))
            {
                __threadfence();
            }
            if (*terminate) {
                return;
            }

            // get the vertex id
            int vertex = warp_vertex_table[wid];

            // color_state
            BitSet color_state;
            bitset_init(&color_state);
            BitSet color;
            bitset_init(&color);

            // local vertex buffer
            int local_vertex[INVALID_MAX_SIZE];
            int invalid_size = 0;

            // traverse the neighbors
            for (int i = offsets[vertex]; i < offsets[vertex + 1]; ++i) {
                int neighbor = adj_list[i];
                if (neighbor > vertex)
                    continue;

                // check if data is valid
                if (conflict_table[neighbor] == true)
                {
                    // get the color of the neighbor
                    BitSet neighbor_color = result[neighbor];

                    // bitwise OR
                    bitset_or(&color_state, &neighbor_color);
                }
                else
                {
                    local_vertex[invalid_size] = neighbor;
                    invalid_size++;
                }
            }
            while (true) {
                bool all_valid = true;
                for (int i = 0; i < invalid_size; ++i) {
                    if (i >= INVALID_MAX_SIZE)
                    {
                        printf("Invalid size exceeded\n");
                        break;
                    }
                    if (local_vertex[i] == -1)
                        continue;
                    if (conflict_table[local_vertex[i]] == true)
                    {
                        // get the color of the neighbor
                        BitSet neighbor_color = result[local_vertex[i]];

                        // bitwise OR
                        bitset_or(&color_state, &neighbor_color);
                        local_vertex[i] = -1;
                    }
                    else
                        all_valid = false;
                }
                if (all_valid)
                    break;
                __threadfence();
            }
            // get the color
            if (bitset_is_zero(&color_state)) {
                bitset_assign_one(&color);
            } else {
                BitSet temp;
                bitset_not(&temp, &color_state);
                bitset_increment(&color_state);
                bitset_and(&color, &temp, &color_state);
            }

            // assign the color to the vertex
            result[vertex] = color;

            // get yourself idle
            atomicExch(&warp_state[wid], IDLE);

            // set dct true
            conflict_table[vertex] = true;
            __threadfence();
            if (*terminate) {
                return;
            }        
        }
    }

    __syncwarp();
}



void free_graph(Graph& graph) {
    delete[] graph.offsets;
    delete[] graph.adj_list;
}

// sort based on the degree of the nodes
// smaller vertex id means higher degree
void sort_adj_list(std::unordered_map<int, std::unordered_set<int>>& adj_list, bool sortOption=true)
{
    std::vector<std::pair<int, int>> degree;
    for (const auto& p : adj_list) {
        degree.push_back({p.first, p.second.size()});
    }

    if (!sortOption)
        return;

    std::cerr << "\nSorting the graph" << std::endl;
    // sort based on the degree, smaller vertex id means higher degree
    // stable sort
    std::sort(degree.begin(), degree.end(), [](const std::pair<int, int>& a, const std::pair<int, int>& b) {
        return a.second > b.second || (a.second == b.second && a.first < b.first);
    });

    // mapping from old vertex id to new vertex id
    std::unordered_map<int, int> mapping;
    for (int i = 0; i < degree.size(); ++i) {
        mapping[degree[i].first] = i;
    }
    
    // remap the vertices
    std::unordered_map<int, std::unordered_set<int>> new_adj_list;
    for (const auto& p : adj_list) {
        int u = mapping[p.first];
        for (int v : p.second) {
            // map the vertex id
            int new_v = mapping[v];
            new_adj_list[u].insert(new_v);

        }
    }

    adj_list = new_adj_list;

    std::cerr << "Sorting done\n" << std::endl;
}

// example format
// 0 1
// 0 2
// 1 2
// 2 3
void read_graph(const std::string& filename, Graph& graph, bool sortOption=true)
{
    std::cerr << "\nReading graph from file: " << filename << std::endl;
    std::ifstream file(filename);
    std::string line;
    std::unordered_map<int, std::unordered_set<int>> adj_list;
    int num_edges = 0;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        int u, v;
        if (!(iss >> u >> v)) {
            break;
        }
        adj_list[u].insert(v);
        adj_list[v].insert(u);
        num_edges += 2;
    }
    file.close();

    // sort the graph
    sort_adj_list(adj_list, sortOption);
        // check if it is starting from 0, if not make it start from 0
    if (!sortOption && adj_list.find(0) == adj_list.end())
    {
        std::cerr << "Vertex 0 not found, remapping the vertices" << std::endl;
        std::unordered_map<int, int> mapping;
        int new_vertex_id = 0;
        for (const auto& p : adj_list) {
            mapping[p.first] = new_vertex_id;
            new_vertex_id++;
        }

        // remap the vertices
        std::unordered_map<int, std::unordered_set<int>> new_adj_list;
        for (const auto& p : adj_list) {
            int u = mapping[p.first];
            for (int v : p.second) {
                // map the vertex id
                int new_v = mapping[v];
                new_adj_list[u].insert(new_v);

            }
        }

        adj_list = new_adj_list;
    }

    graph.num_nodes = adj_list.size();
    graph.num_edges = num_edges;
    graph.offsets = new int[graph.num_nodes + 1];
    graph.adj_list = new int[num_edges];
    int offset = 0;
    graph.offsets[0] = 0;
    for (int i = 0; i < graph.num_nodes; ++i) {
        graph.offsets[i + 1] = graph.offsets[i] + adj_list[i].size();
        for (int v : adj_list[i]) {
            graph.adj_list[offset++] = v;
        }
    }

    std::cerr << "Reading done\n" << std::endl;
}

void print_graph(const Graph& graph) {

    std::cerr << "Number of nodes: " << graph.num_nodes << std::endl;
    std::cerr << "Number of edges: " << graph.num_edges << std::endl;

    // vertex and its neighbors
    for (int i = 0; i < graph.num_nodes; ++i) {
        std::cerr << i << ": ";
        for (int j = graph.offsets[i]; j < graph.offsets[i + 1]; ++j) {
            std::cerr << graph.adj_list[j] << " ";
        }
        std::cerr << std::endl;
    }
}

int gpu_info()
{
    // print total number of cores
    int num_cores = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    // maximum number of streaming multiprocessors
    num_cores = prop.multiProcessorCount;
    std::cerr << "Number of cores: " << num_cores << std::endl;

    // maximum number of threads per block
    int max_threads = prop.maxThreadsPerBlock;
    std::cerr << "Maximum number of threads per block: " << max_threads << std::endl;

    // number of warps, threads per block / warp size * number of cores
    int num_warps = max_threads / 32 * num_cores;
    std::cerr << "Number of warps: " << num_warps << std::endl;

    return num_warps;
}

bool compare_bitset(const BitSet& a, const BitSet& b) 
{
    for (int i = 0; i < MAX_BITS / BITS_PER_WORD; ++i) {
        if (a.bits[i] != b.bits[i]) {
            return false;
        }
    }
    return true;
}

// bitset get host
int bitset_get_host(const BitSet& bs, int index) 
{
    if (index >= MAX_BITS) return 0;
    return (bs.bits[index / BITS_PER_WORD] & (1U << (index % BITS_PER_WORD))) != 0;
}

std::string bitset_get_str(const BitSet& bs) 
{
    std::string result;
    // iterate through the bits
    for (int i = MAX_BITS - 1; i >= 0; --i) {
        result += std::to_string(bitset_get_host(bs, i));
    }

    return result;

}

int check_conflict(const Graph& graph, const BitSet* result) 
{
    int num_conflicts = 0;
    for (int i = 0; i < graph.num_nodes; ++i) {
        for (int j = graph.offsets[i]; j < graph.offsets[i + 1]; ++j) {
            if (compare_bitset(result[i], result[graph.adj_list[j]])) {
                // std::cerr << "Conflict: " << i << " " << graph.adj_list[j] << std::endl;
                num_conflicts++;
            }
        }
    }
    std::cerr << "Number of conflicts: " << num_conflicts << std::endl;
    return num_conflicts;
}

void count_colors(const Graph& graph, const BitSet* result) 
{
    std::unordered_set<std::string> colors;
    for (int i = 0; i < graph.num_nodes; ++i) {
        colors.insert(bitset_get_str(result[i]));
    }
    std::cerr << "Number of colors: " << colors.size() << std::endl;
}
void gpu_process(const Graph& graph, int num_blocks = 30, int block_size = 1024, std::string dataset_name = "dataset")
{
    // allocate the result on the device
    // vertex, color
    BitSet* result;

    // store graph on the device
    int* offsets;
    int* adj_list;

    // store queue on the device
    // it is a queue of vertices, from 0 to num_nodes - 1
    int* queue = new int[graph.num_nodes];
    for (int i = 0; i < graph.num_nodes; ++i) {
        queue[i] = i;
    }

    // copy the result back to the host
    BitSet* result_host = new BitSet[graph.num_nodes];

    // measure time
    hipEvent_t start, stop;
    // launch the kernel
    std::cerr << "Launching kernel" << std::endl;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipMalloc(&result, graph.num_nodes * sizeof(BitSet));
    cudaCheckError();


    hipMalloc(&offsets, (graph.num_nodes + 1) * sizeof(int));
    hipMalloc(&adj_list, graph.num_edges * sizeof(int));
    hipMemcpy(offsets, graph.offsets, (graph.num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(adj_list, graph.adj_list, graph.num_edges * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError();

    int* d_queue;
    hipMalloc(&d_queue, graph.num_nodes * sizeof(int));
    hipMemcpy(d_queue, queue, graph.num_nodes * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError();

    // store number of vertices on the device
    int* num_vertices;
    hipMalloc(&num_vertices, sizeof(int));
    hipMemcpy(num_vertices, &graph.num_nodes, sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError();

    // create data conflict table
    // <vertex, valid> map
    bool* conflict_table;
    // initialize to true
    hipMalloc(&conflict_table, graph.num_nodes * sizeof(bool));
    hipMemset(conflict_table, 1, graph.num_nodes * sizeof(bool));
    cudaCheckError();

    // get the number of warps
    // int num_warps = gpu_info();
    // gpu_info();
    // int num_blocks = 30;
    // int block_size = 1024;
    int num_warps = num_blocks * block_size / 32;

    // create warp state table
    int* warp_state;
    hipMalloc(&warp_state, num_warps * sizeof(int));
    // initialize to IDLE
    hipMemset(warp_state, IDLE, num_warps * sizeof(int));
    cudaCheckError();

    // store number of warps on the device
    int* num_warps_device;
    hipMalloc(&num_warps_device, sizeof(int));
    hipMemcpy(num_warps_device, &num_warps, sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError();

    // store terminate flag on the device
    bool* terminate;
    hipMalloc(&terminate, sizeof(bool));
    // initialize to false
    hipMemset(terminate, 0, sizeof(bool));
    cudaCheckError();

    // create table for <warp, vertex> mapping
    int* warp_vertex_table;
    hipMalloc(&warp_vertex_table, num_warps * sizeof(int));
    // initialize to -1
    hipMemset(warp_vertex_table, -1, num_warps * sizeof(int));
    cudaCheckError();

    kernel<<<num_blocks, block_size>>>(offsets, adj_list, result, d_queue, conflict_table, warp_state, num_vertices, num_warps_device, terminate, warp_vertex_table);
    cudaCheckError();
    hipDeviceSynchronize();
    hipMemcpy(result_host, result, graph.num_nodes * sizeof(BitSet), hipMemcpyDeviceToHost);
    cudaCheckError();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cerr << "Time: " << milliseconds << " ms" << std::endl;


    // check if there is a conflict
    std::cerr << "\nChecking conflicts" << std::endl;
    int conflict = check_conflict(graph, result_host);

    // count the number of colors
    std::cerr << "\nCounting colors" << std::endl;
    count_colors(graph, result_host);

    // free the memory
    delete[] result_host;
    delete[] queue;
    hipFree(result);
    hipFree(offsets);
    hipFree(adj_list);
    hipFree(d_queue);
    hipFree(conflict_table);
    hipFree(warp_state);
    hipFree(num_vertices);
    hipFree(num_warps_device);
    hipFree(terminate);
    hipFree(warp_vertex_table);


    // write result to file as: dataset_name num_blocks block_size time conflict
    std::ofstream file("results.txt", std::ios_base::app);
    file << dataset_name << " " << num_blocks << " " << block_size << " " << milliseconds << " " << conflict << std::endl;
    file.close();



}
void run_test() {
    std::vector<std::string> filenames = {"../datasets/EF.txt", "../datasets/CD.txt", "../datasets/RC.txt", 
                                         "../datasets/CA.txt", "../datasets/RP.txt", "../datasets/RT.txt",
                                         "../datasets/CL.txt"};

    std::vector<int> num_blocks = {1, 8, 16, 30, 1, 1, 1};
    std::vector<int> block_size = {1024, 1024, 1024, 1024, 512, 256, 128};

    for (int i = 0; i < filenames.size(); ++i) {
        for (int k = 0; k < num_blocks.size(); ++k) {
            for (int j = 0; j < TEST_NUM; ++j) {
                std::cerr << "\n\nDataset: " << filenames[i] << " Test: " << j << " Blocks: " << num_blocks[k] << " Block size: " << block_size[k] << std::endl;
                Graph graph;
                read_graph(filenames[i], graph);
                gpu_process(graph, num_blocks[k], block_size[k], filenames[i]);
                free_graph(graph);
            }
        }
    }
}

int main(int argc, char* argv[]) {
    if (argc == 2 && std::string(argv[1]) == "run_tests") {
        run_test();
        return 0;
    }

    if (argc == 2 && std::string(argv[1]) == "gpu_info") {
        gpu_info();
        return 0;
    }

    if (argc == 2 && std::string(argv[1]) == "-h") {
        std::cerr << "Format: " << argv[0] << " <dataset_path> <number_of_blocks> <block_size> <sort_option>" << std::endl;
        std::cerr << "sort_option: sort_yes or sort_no" << std::endl;
        return 0;
    }

    if (argc != 5) {
        std::cerr << "Usage: " << argv[0] << " <dataset_path> <number_of_blocks> <block_size> <sort_option>" << std::endl;
        std::cerr << "Usage 2: " << argv[0] << " run_tests" << std::endl;
        std::cerr << "Usage 3: " << argv[0] << " gpu_info" << std::endl;
        std::cerr << "Usage 4: " << argv[0] << " -h" << std::endl;
        return 1;
    }

    std::string dataset_path = argv[1];
    int num_blocks = std::stoi(argv[2]);
    int block_size = std::stoi(argv[3]);
    
    // sort_option -> sort_yes or sort_no arguments
    bool sort_option = std::string(argv[4]) == "sort_yes";

    if (block_size % 32 != 0 || block_size > 1024 || block_size < 64) {
        std::cerr << "Error: block_size must be a multiple of 32, between 64 and 1024" << std::endl;
        return 1;
    }

    Graph graph;
    read_graph(dataset_path, graph, sort_option);
    gpu_process(graph, num_blocks, block_size, dataset_path);
    free_graph(graph);

    return 0;
}